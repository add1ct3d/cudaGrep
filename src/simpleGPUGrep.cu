//
//  simpleGPUGrep.cu
//  simpleGPUGrep
//
//  Created by HaoJi on 10/29/13.
//  Copyright (c) 2013 HaoJi. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define CHECK_ERR(x)                                    \
  if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
            hipGetErrorString(err),__FILE__,__LINE__); \
    exit(-1);                                           \
  }

#define BLOCKS 100
#define THREADS 1024
#define BUFSIZE 5000
#define FAILURE -1
#define SUCCESS 0

__device__ char *mystrstr(const char *s1, const char *s2) {

	int n;
	if (*s2) {
		while (*s1) {
			for (n = 0; *(s1 + n) == *(s2 + n); n++) {
				if (!*(s2 + n + 1))
					return (char *) s1;
			}
			s1++;
		}
		return NULL ;
	} else
		return (char *) s1;
}

__device__ char *mystrncpy(char *dest, char *source, size_t n) {

	int i;
	if (dest == NULL || source == NULL )
		return NULL ;
	for (i = 0; i < n && source[i] != '\0'; i++) {
		dest[i] = source[i];
	}
	dest[i] = '\0';
	return dest;
}

__device__ int mystrlen(char *str) {

	if (str == NULL )
		return 0;
	int len = 0;
	for (; *str++ != '\0';) {
		len++;
	}
	return len;
}

__global__ void match(char *d_pattern, char* d_lines) {

	//int i = threadIdx.x;
	int pos = blockIdx.x * blockDim.x + threadIdx.x;

	int offset = pos * BUFSIZE;
	char *line = d_lines + offset;
	char *pch = mystrstr(line, d_pattern) != NULL ? line : NULL;

	if (pch != NULL ) {

		//mystrncpy(d_buf + offset, pch, mystrlen(line));
		//mystrncpy(d_buf + offset, pch, mystrlen(line));
		printf("%s", pch);
	}
}

int main(int argc, char *argv[]) {

	hipError_t err;

	char *line;
	char *lines;

	// Memory allocation for pattern, filename (in the host)
	char pattern[BUFSIZE];
	char file_name[BUFSIZE];
	char *d_pattern, *d_lines;

	// Obtain two argv: pattern and file_name
	strcpy(pattern, argv[1]);
	strcpy(file_name, argv[2]);

	// Memory allocation for d_pattern, d_lines (in the device)
	err = hipMalloc((void **) &d_pattern, BUFSIZE);
	CHECK_ERR(err);
	err = hipMalloc((void **) &d_lines, BLOCKS * THREADS * BUFSIZE);
	CHECK_ERR(err);

	// Copying memory to device
	err = hipMemcpy(d_pattern, pattern, BUFSIZE, hipMemcpyHostToDevice);
	CHECK_ERR(err);

	// Memory allocation for lines
	lines = (char*) calloc(BLOCKS * THREADS * BUFSIZE, sizeof(char));

	// Open file
	FILE *fp;
	fp = (FILE *) fopen(file_name, "r");
	if (fp == NULL ) {
		perror("fopen():");
		exit(1);
	}

	// Memory allocation for line
	line = (char*) calloc(BUFSIZE, sizeof(char));

	// n_lines to detect the number of lines in the file
	int n_lines = 0;
	while (fgets(line, BUFSIZE, fp) != NULL ) {

		if (n_lines <= BLOCKS * THREADS - 1) {

			// Copying line to lines
			int offset = n_lines * BUFSIZE;
			strncpy(lines + offset, line, strlen(line));
			memset(line, 0, BUFSIZE);
			n_lines++;

			// Situation that the number of liens in the file is 1024 times
			if (n_lines == BLOCKS * THREADS - 1) {

				// Copying memory to device
				err = hipMemcpy(d_lines, lines, BLOCKS * THREADS * BUFSIZE,
						hipMemcpyHostToDevice);
				CHECK_ERR(err);

				// Calling the kernel
				match<<<BLOCKS, THREADS>>>(d_pattern, d_lines);

				// Reset lines
				n_lines = 0;
				memset(lines, 0, BLOCKS * THREADS * BUFSIZE);
			}
		}
	}

	// Situation that the number of lines in the file not 1024 times
	if (n_lines != 0) {

		// Copying memory to device
		err = hipMemcpy(d_lines, lines, BLOCKS * THREADS * BUFSIZE, hipMemcpyHostToDevice);
		CHECK_ERR(err);

		// Calling the kernel
		match<<<BLOCKS, THREADS>>>(d_pattern, d_lines);
	}

	// Free memory and close file
	free(line);
	free(lines);
	hipFree(d_pattern);
	hipFree(d_lines);
	fclose(fp);

	return SUCCESS;
}
